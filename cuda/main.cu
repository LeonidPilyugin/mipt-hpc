
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// element of matrix
#define INDEX(r, c, rows, columns) ((c) + (r) * (columns))
// element of transposed matrix
#define TINDEX(r, c, rows, columns) ((r) + (c) * (rows))


// keep dimensions multiple to BLOCK
#define X 8192
#define Y 8192
#define Z 8192

#define BLOCK 32

#define BX blockIdx.x
#define BY blockIdx.y
#define TX threadIdx.x
#define TY threadIdx.y

/*
 * left -- left matrix
 * right -- transposed right matrix
 */
__global__ void global_matrix_multiply(double * left, double * right, int3 size, double * result) {
    int rr = BX * BLOCK + TX;
    int rc = BY * BLOCK + TY;
    double sum = 0.0;
    for (int i = 0; i < size.x; i++) {
        sum += left[INDEX(rr, i, size.y, size.x)] * right[INDEX(i, rc, size.x, size.z)];
    }
    result[INDEX(rr, rc, size.y, size.z)] = sum;
}

__global__ void shared_matrix_multiply(double * left, double * right, int3 size, double * result) {
    __shared__ double s_left[BLOCK][BLOCK];
    __shared__ double s_right[BLOCK][BLOCK];

    int rr = BX * BLOCK + TX;
    int rc = BY * BLOCK + TY;
    double sum = 0.0;

    for (int i = 0; i < size.x; i += BLOCK) {
        // load chunks to shared memory
        s_left[TY][TX] = left[INDEX(rr, TY + i, size.y, size.x)];
        s_right[TY][TX] = right[INDEX(TX + i, rc, size.x, size.z)];
        __syncthreads();

        // compute sum
        for (int j = 0; j < BLOCK; j++)
            sum += s_left[j][TX] * s_right[TY][j];

        __syncthreads();
    }

    result[INDEX(rr, rc, size.y, size.z)] = sum;
}

double * generate_matrix(int rows, int columns, int seed=0, double min=0.0, double max=5.0) {
    double * result;
    #ifdef PINNED
    cudaMallocHost((void **) &result, rows * columns * sizeof(double));
    #else
    result = (double *) malloc(rows * columns * sizeof(double));
    #endif

    srand(seed);

    for (int i = 0; i < rows * columns; i++)
        result[i] = min + rand() * (max - min) / ((double) RAND_MAX);

    return result;
}

void print_matrix(double * m, int rows, int columns, FILE * fp) {
    for (int row = 0; row < rows; row++) {
        for (int column = 0; column < columns; column++) {
            fprintf(fp, "%lf ", m[INDEX(row, column, rows, columns)]);
        }
        fprintf(fp, "\n");
    }
}

void destroy_matrix(double * m) {
    #ifdef PINNED
    cudaFreeHost(m);
    #else
    free(m);
    #endif
}

#ifdef SHARED
#define DEVICE_MULTIPLY shared_matrix_multiply
#else
#define DEVICE_MULTIPLY global_matrix_multiply
#endif

#define FOREACH_STREAM(n) for (int i = 0; i < (n); i++)

//#define STREAMS 2

void host_multiply(double * left, double * right, int3 size, double * result) {
    double * dl, * dr, * dres;
    hipMalloc((void **) &dl, size.x * size.y * sizeof(double));
    hipMalloc((void **) &dr, size.x * size.z * sizeof(double));
    hipMalloc((void **) &dres, size.y * size.z * sizeof(double));

    hipMemcpy(dr, right, sizeof(double) * size.x * size.z, hipMemcpyHostToDevice);

    #ifdef STREAMS
    cudaStream_t streams[STREAMS];
    FOREACH_STREAM(STREAMS) cudaStreamCreate(streams + i);

    int3 new_size = make_int3(size.x, size.y / STREAMS, size.z);
    
    FOREACH_STREAM(STREAMS) cudaMemcpyAsync(dl + i * size.x * new_size.y, left + i * size.x * new_size.y, size.x * new_size.y * sizeof(double), cudaMemcpyHostToDevice, streams[i]);

    FOREACH_STREAM(STREAMS) 
    DEVICE_MULTIPLY<<<dim3(new_size.y / BLOCK, size.z / BLOCK), dim3(BLOCK, BLOCK), 0, streams[i]>>>(dl + i * size.x * new_size.y, dr, new_size, dres + i * size.z * new_size.y);

    FOREACH_STREAM(STREAMS) cudaMemcpyAsync(result + i * size.z * new_size.y, dres + i * size.z * new_size.y, size.z * new_size.y * sizeof(double), cudaMemcpyDeviceToHost, streams[i]);

    cudaDeviceSynchronize();

    FOREACH_STREAM(STREAMS) cudaStreamDestroy(streams[i]);
    #else
    hipMemcpy(dl, left, sizeof(double) * size.x * size.y, hipMemcpyHostToDevice);
    DEVICE_MULTIPLY<<<dim3(size.y / BLOCK, size.z / BLOCK), dim3(BLOCK, BLOCK)>>>(dl, dr, size, dres);

    hipMemcpy(result, dres, sizeof(double) * size.z * size.y, hipMemcpyDeviceToHost);
    #endif

    hipFree(dl);
    hipFree(dr);
    hipFree(dres);
}

int main(int argc, char * argv[]) {
    int3 size = make_int3(X, Y, Z);
    double * A = generate_matrix(size.y, size.x, 1);
    double * B = generate_matrix(size.x, size.z, 2);
    double * C = generate_matrix(size.y, size.z, 3);

    clock_t time = clock();
    host_multiply(A, B, size, C);
    time = clock() - time;

    printf("Elapsed time: %lf ms\n", ((double) time) / CLOCKS_PER_SEC * 1000.0);

    FILE * f = fopen("A.matrix", "w");
    print_matrix(A, size.y, size.x, f);
    fclose(f);

    f = fopen("B.matrix", "w");
    print_matrix(B, size.x, size.z, f);
    fclose(f);

    f = fopen("C.matrix", "w");
    print_matrix(C, size.y, size.z, f);
    fclose(f);

    destroy_matrix(A);
    destroy_matrix(B);
    destroy_matrix(C);

    return 0;
}

